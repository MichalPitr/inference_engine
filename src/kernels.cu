#include <hip/hip_runtime.h>

#include <iostream>

__global__ void gemm_kernel(const float *A, const float *B, const float *bias,
                            float *out, int n, int m, int k, bool transA,
                            bool transB, float alpha, float beta) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < k) {
        float res = 0.0f;

        for (int i = 0; i < m; ++i) {
            float aVal = transA ? A[i * n + row] : A[row * m + i];
            float bVal = transB ? B[col * m + i] : B[i * k + col];
            res += aVal * bVal;
        }
        out[row * k + col] = res * alpha + bias[col] * beta;
    }
}

void gemm_cuda(const float *A, const float *B, const float *bias, float *out,
               int n, int m, int k, bool transA, bool transB, float alpha,
               float beta) {
    dim3 blockSize(16, 16);
    dim3 gridSize((k + blockSize.x - 1) / blockSize.x,
                  (n + blockSize.y - 1) / blockSize.y);

    gemm_kernel<<<gridSize, blockSize>>>(A, B, bias, out, n, m, k, transA,
                                         transB, alpha, beta);
}

__global__ void relu_kernel(const float *in, float *out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = in[idx] < 0 ? 0 : in[idx];
    }
}

void relu_cuda(const float *in, float *out, int n) {
    relu_kernel<<<ceil(n / 32.0), 32>>>(in, out, n);
}

__global__ void add_kernel(const float *A, const float *B, float *out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = A[idx] + B[idx];
    }
}

void add_cuda(const float *A, const float *B, float *out, int n) {
    add_kernel<<<ceil(n / 32.0), 32>>>(A, B, out, n);
}